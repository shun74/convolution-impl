#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

__global__ void conv2d_kernel(const uint8_t *pad_src, uint8_t *dst, const float *kernel,
                              int h, int w, int ch, int kh, int kw, int pad_width)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= w * h * ch)
        return;

    int x = tid % w;
    int y = (tid / w) % h;
    int c = tid / (w * h);

    int dst_idx = (y * w + x) * ch + c;

    float sum = 0.0f;
    for (int ky = 0; ky < kh; ky++)
    {
        for (int kx = 0; kx < kw; kx++)
        {
            int py = y + ky;
            int px = x + kx;
            int pad_idx = (py * pad_width + px) * ch + c;
            sum += pad_src[pad_idx] * kernel[ky * kw + kx];
        }
    }
    dst[dst_idx] = (uint8_t)fminf(255.0f, fmaxf(0.0f, sum));
}

extern "C" void conv2d(uint8_t *h_pad_src, uint8_t *h_dst, float *h_kernel,
                       int h, int w, int ch, int kh, int kw)
{
    int pad_width = w + (kw / 2) * 2;
    size_t src_size = (h + kh - 1) * pad_width * ch * sizeof(uint8_t);
    size_t dst_size = h * w * ch * sizeof(uint8_t);
    size_t kernel_size = kh * kw * sizeof(float);

    uint8_t *d_pad_src, *d_dst;
    float *d_kernel;

    hipMalloc(&d_pad_src, src_size);
    hipMalloc(&d_dst, dst_size);
    hipMalloc(&d_kernel, kernel_size);
    hipMemcpy(d_pad_src, h_pad_src, src_size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, h_kernel, kernel_size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    hipEventRecord(start);

    int block = 128;
    int grid = (w * h *ch + block - 1) / block;
    conv2d_kernel<<<grid, block>>>(d_pad_src, d_dst, d_kernel,
                                  h, w, ch, kh, kw, pad_width);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("  execution time: %.3f ms\n", milliseconds);

    hipMemcpy(h_dst, d_dst, dst_size, hipMemcpyDeviceToHost);

    hipFree(d_pad_src);
    hipFree(d_dst);
    hipFree(d_kernel);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

extern "C" void warmup()
{
    hipFree(0);
}
